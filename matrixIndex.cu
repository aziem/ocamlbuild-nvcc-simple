#include <stdio.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <math.h>

void initialData(float *ip, int size) {
  for (int i=0; i < size; i++) {
    ip[i] = (float)rand()/(float)(RAND_MAX/10.0);
  }
}

void print_matrix(float *c, const int nx, const int ny) {
  float *ic = c;
  for (int iy=0; iy<ny; iy++) {
    for (int ix=0; ix<nx; ix++) {
      printf("%6.2f", ic[ix]);
    }
    ic += nx;
    printf("\n");
  }
  printf("\n");
}


__global__ void print_thread_index(float* a, const int nx, const int ny) {
  int ix = threadIdx.x + blockIdx.x * blockDim.x;
  int iy = threadIdx.y + blockIdx.y * blockDim.y;

  unsigned int idx = iy*nx + ix;

  printf("thread_id (%d, %d) block_id (%d, %d) coordinate (%d, %d)"
	 "global index %2d ival %2d\n",
	 threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, ix, iy, idx, a[idx]);
  
}

void test() {
  int dev = 0;
  hipSetDevice(dev);

  int nx = 4;
  int ny = 4;

  int nxy = nx*ny;
  int nbytes = nxy * sizeof(float);

  float *h_A;
  h_A = (float *) malloc(nbytes);

  initialData(h_A, nx*ny);
  print_matrix(h_A, nx, ny);

  float *d_A;
  hipMalloc((void **) &d_A, nbytes);

  hipMemcpy(d_A, h_A, nbytes, hipMemcpyHostToDevice);

  dim3 block(4, 2);
  dim3 grid ((nx+block.x-1)/block.x, (ny+block.y-1)/block.y);
  
  print_thread_index <<< grid, block >>>(d_A, nx, ny);
  hipDeviceSynchronize();

  hipFree(d_A);
  free(h_A);
  hipDeviceReset();
}